#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int main(int argc, char** argv)
{
    // Parse command-line arguments for matrix sizes
    int m = atoi(argv[1]);
    int n = atoi(argv[2]);
    int k = atoi(argv[3]);

    // Initialize cuBLAS library
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Allocate device memory for matrices
    float* d_A, * d_B, * d_C;
    hipMalloc(&d_A, m * k * sizeof(float));
    hipMalloc(&d_B, k * n * sizeof(float));
    hipMalloc(&d_C, m * n * sizeof(float));

    // Allocate host memory for matrices
    float* h_A = (float*)malloc(m * k * sizeof(float));
    float* h_B = (float*)malloc(k * n * sizeof(float));
    float* h_C = (float*)malloc(m * n * sizeof(float));

    // Initialize matrices
    for (int i = 0; i < m * k; i++) h_A[i] = (float)rand() / RAND_MAX;
    for (int i = 0; i < k * n; i++) h_B[i] = (float)rand() / RAND_MAX;
    for (int i = 0; i < m * n; i++) h_C[i] = 0;

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, m * n * sizeof(float), hipMemcpyHostToDevice);

    // Perform matrix multiplication
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B, n, d_A, k, &beta, d_C, n);

    // Copy result matrix from device to host
    hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }

    // Print result matrix
    //printf("Result matrix:\n");
    //for (int i = 0; i < m; i++) {
    //    for (int j = 0; j < n; j++) {
    //        printf("%f ", h_C[j * m + i]);
    //    }
    //    printf("\n");
    //}

    // Clean up resources
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}

